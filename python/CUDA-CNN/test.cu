#include "hip/hip_runtime.h"

 
#include <iostream>
using namespace std;
#define TILE_WIDTH 16  
 
__global__ void MatrixMulKernle(int m, int n, int k, float *A,float  *B, float *C)
{
	 //申请共享内存，存在于每个block中
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
 
	//简化坐标记法,出现下面6个表示的地方就是并行的地方。
	int bx = blockIdx.x;		int by = blockIdx.y;
	int tx = threadIdx.x;		int ty = threadIdx.y;
 
	//确定结果矩阵中的行和列
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
 
	//临时变量
	float Cvalue = 0;
 
	//循环读入A,B瓦片，计算结果矩阵，分阶段进行计算
	for (int t=0; t<(n-1)/TILE_WIDTH+1; ++t)
	{
		//将A,B矩阵瓦片化的结果放入shared memory中，每个线程加载相应于C元素的A/B矩阵元素
		if (Row < m && t * TILE_WIDTH + tx < n)		//越界处理，满足任意大小的矩阵相乘（可选）
			//ds_A[tx][ty] = A[t*TILE_WIDTH + tx][Row];
		    ds_A[tx][ty] = A[Row * n + t * TILE_WIDTH + tx];//以合并的方式加载瓦片
		else
			ds_A[tx][ty] = 0.0;
 
		if (t * TILE_WIDTH + ty < n && Col < k)
			//ds_B[tx][ty] = B[Col][t*TILE_WIDTH + ty];
            ds_B[tx][ty] = B[(t * TILE_WIDTH + ty) * k + Col];
		else
			ds_B[tx][ty] = 0.0;	
 
		//保证tile中所有的元素被加载
		__syncthreads();
		
		for (int i = 0; i < TILE_WIDTH; ++i)
            Cvalue += ds_A[i][ty] * ds_B[tx][i];//从shared memory中取值
 
		//确保所有线程完成计算后，进行下一个阶段的计算
		__syncthreads();
 
		if(Row < m && Col < k)
			C[Row*k+Col]=Cvalue;		
	}
}
 
int main()
{
	//这里将矩阵按照行优先转换成了一维的形式
	//实际中数组数量巨大，这里只是为了演示程序的过程
	float A[6] = { 11.4, 24, 33.5,   45, 55 ,32.4 }; //2×3的矩阵
	float B[12] = {12,43,22.4, 31.3,  12,324,23,12,  44.4,23.4,65.3,73};//3×4的矩阵
	float C[8] = { 0 }; //2×4的结果矩阵
	int m=2,n=3,k=4;
 
	//分配显存空间
	int size = sizeof(float);
	float *d_a;
	float *d_b;
	float *d_c;
	hipMalloc((void**)&d_a,m*n*size);
	hipMalloc((void**)&d_b,n*k*size);
	hipMalloc((void**)&d_c,m*k*size);
 
	//把数据从Host传到Device
	hipMemcpy(d_a, A, size*6, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size*12, hipMemcpyHostToDevice);
	hipMemcpy(d_c, C, size*8, hipMemcpyHostToDevice);
 
	//分配网格结构
	dim3 dimGrid((k-1)/TILE_WIDTH+1,(m-1)/TILE_WIDTH+1,1);	//向上取整
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
 
	//调用内核函数
	MatrixMulKernle<<<dimGrid,dimBlock>>>(m,n,k,d_a,d_b,d_c);
 
	//将结果传回到主机端
	hipMemcpy(C, d_c, size*8, hipMemcpyDeviceToHost);
 
	//输出结果
	for (int i=0;i<8;i++)
	{
		cout<<C[i]<<endl;
	}
 
	//释放显存空间
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
 
	return 0;
}