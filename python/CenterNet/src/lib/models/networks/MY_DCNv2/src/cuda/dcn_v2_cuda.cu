#include <vector>
#include "cuda/dcn_v2_im2col_cuda.h"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

extern THCState *state;


at::Tensor
dcn_v2_cuda_forward(const at::Tensor &input, 
                    const at::Tensor &weight,
                    const at::Tensor &bias,
                    const at::Tensor &offset,
                    const at::Tensor &mask,
                    const int kernel_h,
                    const int kernel_w,
                    const int stride_h,
                    const int stride_w,
                    const int pad_h,
                    const int pad_w,
                    const int dilation_h,
                    const int dilation_w,
                    const int deformable_group)
{
    using scalar_t = float;

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");
    AT_ASSERTM(offset.type().is_cuda(), "offset must be a CUDA tensor");
    AT_ASSERTM(mask.type().is_cuda(), "mask must be a CUDA tensor");
    
    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h = weight.size(2);
    const int kernel_w = weight.size(3);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
        "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == channels_kernel,
        "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel);

}